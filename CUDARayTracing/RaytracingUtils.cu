#include "hip/hip_runtime.h"
#include "RaytracingUtils.h"

#include "EntityList.h"
#include "Shape.h"
#include "Camera.h"
#include "Screen.h"

#define SEED 1984
#define RANDBLOCKSIZE 10000

namespace
{
	__device__ Vector3 background_color(const Ray& ray)
	{
		Vector3 unit_direction = normalize(ray.direction());
		float t = 0.5f*(unit_direction.y() + 1.0f);
		return (1.0f - t)*Vector3(1.0, 1.0, 1.0) + t * Vector3(0.5, 0.7, 1.0);
	}

	__device__ Vector3 color(const Ray& ray, const EntityList** entities, hiprandState* rand)
	{
		Ray propagatedRay = ray;
		Vector3 totalAttenuation = Vector3(1.0f, 1.0f, 1.0f);
		float cudaAtt = 1.0f;

		for (int i = 0; i < 2; ++i)
		{
			HitInfo hitInfo;
			if ((*entities)->Hit(propagatedRay, 0.001f, FLT_MAX, hitInfo))
			{
				ScatterInfo scatterInfo;
				if (hitInfo.entity->Scatter(propagatedRay, hitInfo, scatterInfo, rand))
				{
					totalAttenuation *= scatterInfo.attenuation;
					propagatedRay = scatterInfo.scatteredRay;
				}
				else
				{
					return Vector3(0.0f, 0.0f, 0.0f);
				}
			}
			else
			{
				return totalAttenuation * background_color(propagatedRay);
			}
		}

		return Vector3(0.0f, 0.0f, 0.0f);
	}

	__global__ void render_colors(Screen* screen, EntityList** entities, Camera* camera, hiprandState* randStates, int pixelsWidth, int pixelsHeight)
	{
		const int i = threadIdx.x + blockIdx.x * blockDim.x;
		const int j = threadIdx.y + blockIdx.y * blockDim.y;
		if ((i >= pixelsWidth) || (j >= pixelsHeight)) 
            return;

		const int pixelIndex = j * pixelsWidth + i;
        hiprandState& randState = randStates[pixelIndex];

		const float u = (float(i + hiprand_uniform(&randState))) / float(pixelsWidth);
		const float v = (float(j + hiprand_uniform(&randState))) / float(pixelsHeight);

		screen->AddColor(color(camera->GenerateRay(u, v), entities, &randState), i, j);
	}

	__global__ void init_render(hiprandState* randStates, int pixelsWidth, int pixelsHeight)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		int j = threadIdx.y + blockIdx.y * blockDim.y;
		if ((i >= pixelsWidth) || (j >= pixelsHeight)) 
			return;
		int index = j * pixelsWidth + i;

		hiprand_init((SEED << 20) + index, 0, 0, &randStates[index]);
	}

	__global__ void create_entities(EntityList** entities)
	{
		if (threadIdx.x == 0 && blockIdx.x == 0) 
		{
			*(entities) = new EntityList(5);
			(*entities)->push_back(new Entity(new Sphere(Vector3(0.0f, 0.0f, -1.0f), 0.5f), new Lambertian(Vector3(0.8f, 0.3f, 0.3f))));
			(*entities)->push_back(new Entity(new Sphere(Vector3(0.0f, -100.5f, -1.0f), 100.0f), new Lambertian(Vector3(0.8f, 0.8f, 0.0f))));
			(*entities)->push_back(new Entity(new Sphere(Vector3(1.0f, 0.0f, -1.0f), 0.5f), new Metal(Vector3(0.8f, 0.6f, 0.2f), 1.0f)));
			(*entities)->push_back(new Entity(new Sphere(Vector3(-1.0f, 0.0f, -1.0f), 0.5f), new Dielectric(1.5f)));
			(*entities)->push_back(new Entity(new Sphere(Vector3(-1.0f, 0.0f, -1.0f), -0.45f), new Dielectric(1.5f)));
		}
	}

	__global__ void free_entities(EntityList** entities)
	{
		delete *entities;
	}
}

namespace RaytracingUtils
{
	__host__ void getColors(Screen* screen, EntityList** entities, Camera* camera, hiprandState* randStates, int pixelsWidth, int pixelsHeight, int threadsX, int threadsY)
	{
		dim3 blocks(pixelsWidth / threadsX + 1, pixelsHeight / threadsY + 1);
		dim3 threads(threadsX, threadsY);
		render_colors<<<blocks, threads>>>(screen, entities, camera, randStates, pixelsWidth, pixelsHeight);

		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		screen->AddSample();
	}

    __host__ void initRender(hiprandState* randStates, int pixelsWidth, int pixelsHeight, int threadsX, int threadsY)
    {
		dim3 blocks(pixelsWidth / threadsX + 1, pixelsHeight / threadsY + 1);
		dim3 threads(threadsX, threadsY);
		init_render<<<blocks, threads>>>(randStates, pixelsWidth, pixelsHeight);
    }

	__host__ void initEntities(EntityList** entities)
	{
		create_entities<<<1, 1>>>(entities);
	}

	__host__ void cleanUpEntities(EntityList** entities)
	{
		free_entities<<<1, 1 >>>(entities);
	}
}

namespace MathUtils
{
	__device__ Vector3 RandomPointInSphere(hiprandState* rand)
	{
		Vector3 ret;
		do
		{
			ret = 2.0f * Vector3(hiprand_uniform(rand), hiprand_uniform(rand), hiprand_uniform(rand)) - Vector3(1.0f, 1.0f, 1.0f);
		} while (ret.lengthSq() >= 1.0f);
		return ret;
	}

	__device__ Vector3 ReflectedVector(const Vector3 & inVector, const Vector3 & normal)
	{
		return inVector - 2.0f * dot(inVector, normal) * normal;
	}

	__device__ float CosineIncidentAngle(const Vector3& normal, const Vector3& inVector)
	{
		return -dot(normal, inVector);
	}

	//Snell's law vectorial form
	// v_refract = r v + (r c - sqrt(1 - r^2 (1 - c^2))) n
	// r = n1/n2, c = - n * v
	__device__ bool Refracts(const Vector3& inVector, const Vector3& normal, float refractionFactorRatio, Vector3& refracted)
	{
		float c = CosineIncidentAngle(normal, inVector);
		float discriminant = 1 - refractionFactorRatio * refractionFactorRatio * (1 - c * c);
		if (discriminant < 0)
		{
			//Total internal reflection
			return false;
		}

		refracted = refractionFactorRatio * inVector + (refractionFactorRatio * c - sqrt(discriminant)) * normal;
		return true;
	}

	//Approximates reflection coefficient as function of incident angle
	__device__ float SchlickApproximation(float refractionFactorRatio, float cosine)
	{
		float r0 = (refractionFactorRatio - 1.0f) / (refractionFactorRatio + 1.0f);
		r0 = r0 * r0;
		return r0 + (1.0f - r0) * powf(1.0f - cosine, 5);
	}
}